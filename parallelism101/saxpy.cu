#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main(void)
{
  int N = 1 << 20;
  float *x, *y, *d_x, *d_y;
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, &start);

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);

  clock_gettime(CLOCK_MONOTONIC, &end);
  double time_taken;
  time_taken = (end.tv_sec - start.tv_sec) * 1e9;
  time_taken = (time_taken + (end.tv_nsec - start.tv_nsec)) * 1e-9;
  printf("Time: %f\n", time_taken);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  free(x);
  free(y);
}